#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;
const int N=64;
__global__
void square(unsigned matrix[][N], unsigned result[][N], unsigned N){
	//result = matrix;
	unsigned idx = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned idy = blockIdx.y * blockDim.y + threadIdx.y;
	//unsigned ii = id / N;
	//unsigned jj = id % N;
	for(unsigned kk = 0; kk < N; ++kk) {
		result[idx][idy] += matrix[idx][kk] * matrix[kk][idy];
	}
	
}
__global__
void initArray(unsigned matrix[][N], unsigned N)
{
	unsigned idx = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned idy = blockIdx.y * blockDim.y + threadIdx.y;

	matrix[idx][idy] = (idx * N + idy);
}

int main() {
//int N = 64;
//int m = 64, n = 64;
unsigned matrix[N][N];
unsigned (*gMatrix)[N];
unsigned (*result)[N];
hipMallocManaged(&gMatrix, (N*N)*sizeof(int));
hipMallocManaged(&result, (N*N)*sizeof(int));
int threadCount = 32;


dim3 threadsPerBlock(threadCount, threadCount);
dim3 numBlocks(N / threadsPerBlock.x, N / threadsPerBlock.y);


initArray<<<numBlocks,threadsPerBlock>>>(gMatrix,N);
square<<<numBlocks,threadsPerBlock>>>(gMatrix, result,N);

cout << "Hello world"; //test
hipMemcpy(matrix, result, (N*N)*sizeof(int), hipMemcpyDeviceToHost);;
cout << "Squared matrix = ";
for(int i = 0; i < N; i++){
	cout << endl;
	for(int j = 1; j < N; ++j)
	{
		cout <<matrix[i][j]<< " ";
	}
}
hipFree(result);
hipFree(gMatrix);
}
